#include "hip/hip_runtime.h"
#include "./prefixScan.cuh"
#include <>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error: %s:%d, %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

void check_cumsumprod(const float* A, const float* B, const float* output, int n, float epsilon = 1e-3) {
    printf("Checking for errors...\n");
    
    // For sequential verification, we need to track running sums and products
    float runningSum = 0.0f;  // Equivalent to the combined sum across all elements
    
    for (int i = 0; i < n; ++i) {
        // Calculate the expected value:
        // B[i] (current value) + runningSum (sum of all previous elements)
        runningSum = B[i] + runningSum * A[i];
        
        if ((fabs(output[i] - runningSum) >= epsilon)) {
            printf("Error at index %d: GPU output = %f, Expected = %f\n", i, output[i], runningSum);
            assert(fabs(output[i] - runningSum) < epsilon);
        }
    }
    printf("All results match within epsilon=%f\n", epsilon);
}

void allocateArray(float*& array, int n, float init_value = 0) {
    array = (float*)malloc(n * sizeof(float));
    if (array == NULL) {
        printf("Host memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < n; ++i) {
        array[i] = init_value;
    }
}

int main() {
    int n = 1000000; // the amount of values in the array
    int numRuns = 100;
    int bytes = sizeof(float) * n;
    
    float *h_A, *h_B, *h_output;
    float *d_A, *d_B, *d_output;
    
    // Initialize arrays
    allocateArray(h_A, n, 0.99f);  // Values for multiplication
    allocateArray(h_B, n, 0.5f);   // Values for addition
    allocateArray(h_output, n);
    
    CHECK_CUDA(hipMalloc(&d_A, bytes));
    CHECK_CUDA(hipMalloc(&d_B, bytes));
    CHECK_CUDA(hipMalloc(&d_output, bytes));
    
    CHECK_CUDA(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));
    
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    float totalTime = 0;
    for (int i = 0; i < numRuns; i++) {
        CHECK_CUDA(hipEventRecord(start));
        
        // Call the cumsumprod algorithm
        cumsumprodScan(d_A, d_B, d_output, n);
        
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        
        float milliseconds = 0;
        CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
        totalTime += milliseconds;
    }
    
    printf("Average execution time over %d runs: %f ms\n", numRuns, totalTime / numRuns);
    
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    
    // Verify results
    check_cumsumprod(h_A, h_B, h_output, n);
    
    // Free memory
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_output));
    
    free(h_A);
    free(h_B);
    free(h_output);
    
    printf("CumSumProd completed successfully\n");
    
    return 0;
}