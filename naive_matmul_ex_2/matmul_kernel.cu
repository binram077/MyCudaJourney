#include "hip/hip_runtime.h"
#include "./matmul_kernel.cuh"

__global__ void matmul(float* a, float* b, float* c, int n) {
    int id_x = (blockDim.x * blockIdx.x) + threadIdx.x;
    int id_y = (blockDim.y * blockIdx.y) + threadIdx.y;

    if (id_x < n && id_y < n) {
        float tmp = 0;
        for (int i = 0; i < n; ++i) {
            tmp += a[id_x * n + i] * b[i * n + id_y];
        }
        c[id_x * n + id_y] = tmp;
    }
}